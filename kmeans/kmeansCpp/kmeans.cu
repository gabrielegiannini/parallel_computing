#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iostream>
#include <string>
#include <fstream>
#include <utility>
#include <vector>
#include <unordered_map>
#include <future>
#include <filesystem>
#include <thread>
#include <float.h>

namespace fs = std::filesystem;

using namespace std;

#define N 10000000
#define MAX_ERR 1e-6

#define CLUSTER_NUMBER 5
#define ELEMENTS_NUMBER 10
#define ARRAYSIZEOF(ptr) (sizeof(ptr)/sizeof(ptr[0]))

void kmean(double totalNormAvg[5],  unordered_map<string, double> entry, double means[5]) {
        double norm = 0;
        vector<vector<string>> S(5);// dimensione clusterNumber. S e' array list di array list
        /*for (int j = 0; j < S.length; j++) {
            S[j] = new ArrayList<>();
        }*/
        for (int h = 0; h < 5; h++) {// array delle norme. no cuda
            totalNormAvg[h] = 0;
        }
        for (const auto& e : entry ) {// boh si parrallelizza roba qui? sono tutte assegnazioni e una somma, non so quanto possa valerne la pena. ma soprattutto, come la prende se norm al suo interno e` anche lei parallelizzata?
            int posMin = 0;

            double min = DBL_MAX;
            for (int h = 0; h < 5; h++) {//direi che questo for non importa parallelizzarlo con cuda visto che sono solo assegnazioni apparte norm che pero` e` gia` fatto
                //double norm = norm(entry.getValue(), means[h]);
                if (norm < min) {
                    min = norm;
                    posMin = h;
                }
            }
            S[posMin].push_back(e.first); //è sbagliato era solo per provare
            totalNormAvg[posMin] = totalNormAvg[posMin] + min;
        }
        for (int i = 0; i < 5; i++) {
            if (S[i].size() > 0) {
                totalNormAvg[i] = totalNormAvg[i] / S[i].size();
            }
        }
}


__global__ void normA(double vect[], double centroids[], double res[], int n, double sum[], size_t dataSize) {
    /* 
       Calcoliamo la norma fra un vettore e un centroide
       allora, res contiene i risultati intermedi del calcolo della norma, ovvero i quadrati delle differenze fra coordinate corrispondenti dei vettori
       quindi e' grande #vettori*#cluster*#coordinate(cioe' dimensione dei singoli vettori, cioe' n)
       
       blockIdx.y identifica il vettore di cui calcolare la norma
       blockIdx.x identifica il cluster, ovvero il centroide con cui fare la norma
       threadIdx.x identifica la coordinata di cui si deve occupare il singolo core
    */
    printf("Indice res %lu\n",blockIdx.y*n + blockIdx.x*dataSize + threadIdx.x);
    res[blockIdx.y*n + blockIdx.x*dataSize*n + threadIdx.x] = pow(vect[blockIdx.y*n + threadIdx.x] - centroids[blockIdx.x*n + threadIdx.x], 2);
    __syncthreads();
    if(threadIdx.x == 0){
        for(int i=0; i<n;i++){
            sum[blockIdx.x*dataSize+blockIdx.y] = sum[blockIdx.x*dataSize+blockIdx.y] + res[blockIdx.y*n + blockIdx.x*dataSize*n + i];
        }
    }
}

// dataSize è il numero di vettori, ovvero sizeof(data) / n (sennò aveva davvero poco senso)
__global__ void kmeanDevice(double S[], int dimS[], int * elemLengthPtr, double totalNormAvg[],  double data[], double centroids[], double res[], double sum[], size_t dataSize, size_t clusterNumber){
        int n = *elemLengthPtr;
        
        int *posMin = new int[dataSize];
        double *min = new double[dataSize]; //inizializzare a DBL_MAX

        for (int h = 0; h < dataSize; h++) {// array delle norme. no cuda
            min[h] = DBL_MAX;
            posMin[h] = 0;
        }

        int *filledS = new int[clusterNumber];
        for (int h = 0; h < clusterNumber; h++) {// array delle norme. no cuda
            dimS[h] = 0;
            totalNormAvg[h] = 0;
            filledS[h] = 0;
        }

    //norm(data, means);
    dim3 numBlocks(clusterNumber, dataSize);
    printf("Sto per fare norm\n");
    normA<<<numBlocks,n>>>(data, centroids, res, n, sum, dataSize);
    hipDeviceSynchronize();
    for (int v=0; v<dataSize;v++){
            for (int h = 0; h < clusterNumber; h++) {//direi che questo for non importa parallelizzarlo con cuda visto che sono solo assegnazioni apparte norm che pero` e` gia` fatto

                if (sum[h*dataSize+v] < min[v]) {
                    min[v] = sum[h*dataSize+v];
                    posMin[v] = h;
                }
            }
            dimS[posMin[v]] += 1;
        }

        for (int l = 0; l<dataSize; l++){
            int targetPosition = 0;
            for (int i = 0; i < posMin[l]; i++) {
                targetPosition += dimS[i];
            }
            targetPosition += filledS[posMin[l]];
            for (int k=0;k<n;k++){
                S[targetPosition*n+k] = data[l*n+k];
            }
            filledS[posMin[l]] += 1;
            totalNormAvg[posMin[l]] = totalNormAvg[posMin[l]] + min[l];
        }

        for (int i = 0; i < clusterNumber; i++) {
            if (dimS[i] > 0) {
                totalNormAvg[i] = totalNormAvg[i] / dimS[i];
            }
        }
}

__global__ void meanz(double centroids[], double S[], int dimS[], int * elemLengthPtr) {// calcola centroidi
    int elemLength = *elemLengthPtr;
    centroids[blockIdx.x*elemLength + threadIdx.x] = 0;
    int dimSum = 0;
    // calcola la coordinata iniziale del primo vettore del cluster blockIdx.x
    for (int j=0; j<blockIdx.x; j++) {
        dimSum += dimS[j];
    }
    dimSum = dimSum*elemLength;
    // scorre tutti gli elementi del cluster (la grandezza del cluster e' in dimS[blockIdx.x])
    for (int i=0; i<dimS[blockIdx.x]; i++) {
        dimSum += elemLength;
        // quindi alla fine in centroids c'e' la somma di tutte le n-esime coordinate di ogni elemento del cluster
        centroids[blockIdx.x *elemLength + threadIdx.x] = centroids[blockIdx.x *elemLength + threadIdx.x] + S[dimSum + threadIdx.x];

    }
    // divide per la dimensione del cluster per fare la media -> coordinata n-esima del nuovo centroide di questo cluster
    centroids[blockIdx.x *elemLength + threadIdx.x] = centroids[blockIdx.x *elemLength + threadIdx.x] / dimS[blockIdx.x];
}


unsigned long parseData(ifstream &csv, vector<double> &data) {
        double *domainMax;
        unsigned long n = -1;
        int index = 0;
        while (!csv.eof()) {
            string row;
            getline(csv, row);
            //cout << row << endl;
            istringstream iss(row);
            // perche ovviamente in c++ string.split() non esiste...
            vector<string> rowArr;
            const char delimiter = ';';
            // evita il primo token, tanto è il nome del primo vettore
            int start = row.find(delimiter) + 1;
            int end = row.find(delimiter, start);
            while (end != -1) {
                rowArr.push_back(row.substr(start, end - start));
                start = end + 1; // scansa il ';'
                end = row.find(delimiter, start);
            }
            rowArr.push_back(row.substr(start));

            if (n == -1) {
                n = rowArr.size();
                domainMax = new double[n];
                for (int i = 0; i < n; i++) {
                    domainMax[i] = std::numeric_limits<double>::lowest();
                }
            }
            if (n == rowArr.size())
            {
                for (int i = 0; i < n; i++)
                {
                    data.push_back(stod(rowArr[i]));
                    if (data[index] > domainMax[i])
                    {
                        domainMax[i] = data[index];
                    }
                    index++;
                }
            }
        }
        for(int j=0; j<data.size(); j++) {
            data[j] = data[j] / domainMax[j%n]; // normalizza i dati -> tutto è adesso fra 0 e 1
        }
        return n;
    }

int main(){
    double *a, *b, *out;
    double *d_a, *d_b, *d_out;
    double *res;
    double *sum;
    double *sum_host;
    double *means;
    double *S;
    double *S_host;
    int *dimS;
    int *dimS_host;
    int *elemLength;
    double *totalNormAvg;
    double *centroids;
    double *data_d;

    vector<double> dataVec(0);
    //vector<double> totalNormAvg(CLUSTER_NUMBER);
    ifstream myfile;
    //myfile.open("../../datasetProva.csv");
    myfile.open("../../test_reale.csv");
    unsigned long n = parseData(myfile, dataVec);
    myfile.close();
    double data[dataVec.size()];
    std::copy(dataVec.begin(), dataVec.end(), data);
    //printf(dataVec.size());
    cout << "n = " << n << "\n";
    cout << "Datavec size: " << dataVec.size() << endl;
    cout << "Data size: " << ARRAYSIZEOF(data) << endl;

    // Allocate host memory
    a   = (double*)malloc(sizeof(double) * N);
    b   = (double*)malloc(sizeof(double) * N);
    out = (double*)malloc(sizeof(double) * N);
    S_host=(double*)malloc(sizeof(double) * dataVec.size());
    dimS_host=(int*)malloc(sizeof(int) * CLUSTER_NUMBER);
    sum_host = (double*)malloc(sizeof(double) * dataVec.size()/n*CLUSTER_NUMBER);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&res, sizeof(double) * dataVec.size()*CLUSTER_NUMBER);
    hipMalloc((void**)&sum, sizeof(double) * dataVec.size()/n*CLUSTER_NUMBER);
    hipMalloc((void**)&S, sizeof(double) * dataVec.size());
    hipMalloc((void**)&dimS, sizeof(double) * CLUSTER_NUMBER);
    hipMalloc((void**)&totalNormAvg, sizeof(double) * CLUSTER_NUMBER);
    hipMalloc((void**)&elemLength, sizeof(int));
    hipMalloc((void**)&centroids, sizeof(double) * CLUSTER_NUMBER*n);
    hipMalloc((void**)&data_d, sizeof(double) * dataVec.size());


    hipMalloc((void**)&d_a, sizeof(double) * N);
    hipMalloc((void**)&d_b, sizeof(double) * N);
    hipMalloc((void**)&d_out, sizeof(double) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(data_d, data, sizeof(double) * dataVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(elemLength, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(centroids, data, sizeof(double)*n*CLUSTER_NUMBER, hipMemcpyHostToDevice); //i primi CLUSTER_NUMBER vettori di data per provare

    // Executing kernel 
    //normA<<<CLUSTER_NUMBER,5>>>(d_a, d_b, res, 5, sum);

    kmeanDevice<<<1,1>>>(S, dimS, elemLength, totalNormAvg,  data_d, centroids, res, sum, ARRAYSIZEOF(data)/n, CLUSTER_NUMBER);
    //hipDeviceSynchronize();
    //meanz<<<CLUSTER_NUMBER, 5>>>(means, S, dimS, elemLength);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipMemcpy(S_host, S, sizeof(double) * dataVec.size(), hipMemcpyDeviceToHost);
    int index = 1;
    for(int i = index*n; i<(index+1)*n; i++){
        cout << S_host[i] << endl;
    }
    cout << "\n";
    hipMemcpy(dimS_host, dimS, sizeof(int) * CLUSTER_NUMBER, hipMemcpyDeviceToHost);
    for(int i = 0; i<CLUSTER_NUMBER; i++){
        cout << dimS_host[i] << endl;
    }
    cout << "\n";
    hipMemcpy(sum_host, sum, sizeof(double) * dataVec.size()/n*CLUSTER_NUMBER, hipMemcpyDeviceToHost);
    for(int i = 0; i<20; i++){
        cout << sum_host[i] << endl;
    }
    dim3 numBlocks(CLUSTER_NUMBER, ARRAYSIZEOF(data));
    cout << "Dimensione grid: " << CLUSTER_NUMBER << "x" << ARRAYSIZEOF(data)/n << endl;

    // Verification

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a);
    free(b);
    free(out);

    cout << "Esecuzione terminata." << endl;
}


