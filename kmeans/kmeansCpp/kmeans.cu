#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iostream>
#include <string>
#include <fstream>
#include <utility>
#include <vector>
#include <unordered_map>
#include <future>
#include <filesystem>
#include <thread>
#include <cfloat>
#include <sstream>
#include <iomanip>

namespace fs = std::filesystem;

using namespace std;

#define N 10000000
#define MAX_ERR 1e-6

#define CLUSTER_NUMBER 5
#define ARRAYSIZEOF(ptr) (sizeof(ptr)/sizeof(ptr[0]))

__global__ void normA(const double vect[], const double centroids[], double res[], size_t n, double sum[], size_t dataSize) {
    /* 
       Calcoliamo la norma fra un vettore e un centroide
       allora, res contiene i risultati intermedi del calcolo della norma, ovvero i quadrati delle differenze fra coordinate corrispondenti dei vettori
       quindi e' grande #vettori*#cluster*#coordinate(cioe' dimensione dei singoli vettori, cioe' n)
       
       blockIdx.y identifica il vettore di cui calcolare la norma
       blockIdx.x identifica il cluster, ovvero il centroide con cui fare la norma
       threadIdx.x identifica la coordinata di cui si deve occupare il singolo core
    */
    //printf("Indice res %lu\n",blockIdx.y*n + blockIdx.x*dataSize*n + threadIdx.x);
    res[blockIdx.y*n + blockIdx.x*dataSize*n + threadIdx.x] = pow(vect[blockIdx.y*n + threadIdx.x] - centroids[blockIdx.x*n + threadIdx.x], 2);
    __syncthreads();
    if(threadIdx.x == 0){
        for(int i=0; i<n;i++){
            sum[blockIdx.x*dataSize+blockIdx.y] = sum[blockIdx.x*dataSize+blockIdx.y] + res[blockIdx.y*n + blockIdx.x*dataSize*n + i];
        }
    }
}

// dataSize è il numero di vettori, ovvero sizeof(data) / n (sennò aveva davvero poco senso)
__global__ void kmeanDevice(int S[], int dimS[], size_t n, double totalNormAvg[],  const double data[], double centroids[], double res[], double sum[], size_t dataSize, size_t clusterNumber){
    int *posMin = new int[dataSize];
    auto *min = new double[dataSize]; //inizializzare a DBL_MAX

    for (int h = 0; h < dataSize; h++) {// array delle norme. no cuda
        min[h] = DBL_MAX;
        posMin[h] = 0;
    }

    int *filledS = new int[clusterNumber];
    for (int h = 0; h < clusterNumber; h++) {// array delle norme. no cuda
        dimS[h] = 0;
        totalNormAvg[h] = 0;
        filledS[h] = 0;
    }

    //norm(data, means);
    dim3 numBlocks(clusterNumber, dataSize);
    //printf("Sto per fare norm\n");
    normA<<<numBlocks,n>>>(data, centroids, res, n, sum, dataSize);
    hipDeviceSynchronize();
    for (int v=0; v<dataSize;v++){
        for (int h = 0; h < clusterNumber; h++) {//direi che questo for non importa parallelizzarlo con cuda visto che sono solo assegnazioni apparte norm che pero` e` gia` fatto
            if (sum[h*dataSize+v] < min[v]) {
                min[v] = sum[h*dataSize+v];
                posMin[v] = h;
            }
        }
        dimS[posMin[v]] += 1;
    }

    for (int l = 0; l<dataSize; l++){
        int targetPosition = 0;
        for (int i = 0; i < posMin[l]; i++) {
            targetPosition += dimS[i];
        }
        targetPosition += filledS[posMin[l]];
//        for (int k=0;k<n;k++){
//            S[targetPosition*n+k] = data[l*n+k];
//        }
        S[targetPosition] = l;
        filledS[posMin[l]] += 1;
        totalNormAvg[posMin[l]] = totalNormAvg[posMin[l]] + min[l];
    }

    for (int i = 0; i < clusterNumber; i++) {
        if (dimS[i] > 0) {
            totalNormAvg[i] = totalNormAvg[i] / dimS[i];
        }
    }
    delete[] filledS;
    delete[] min;
    delete[] posMin;
}

__global__ void meanz(double centroids[], const double data[], const int S[], const int dimS[], size_t n) {// calcola centroidi
    centroids[blockIdx.x * n + threadIdx.x] = 0;
    size_t dimSum = 0;
    // calcola la coordinata iniziale del primo vettore del cluster blockIdx.x
    for (int j=0; j<blockIdx.x; j++) {
        dimSum += dimS[j];
    }
//    dimSum = dimSum * n;
    // scorre tutti gli elementi del cluster (la grandezza del cluster e' in dimS[blockIdx.x])
    for (int i=0; i<dimS[blockIdx.x]; i++) {
        //dimSum += n;
        // quindi alla fine in centroids c'e' la somma di tutte le n-esime coordinate di ogni elemento del cluster
        centroids[blockIdx.x * n + threadIdx.x] = centroids[blockIdx.x * n + threadIdx.x] + data[S[dimSum]*n + threadIdx.x];
        dimSum += 1;

    }
    // divide per la dimensione del cluster per fare la media -> coordinata n-esima del nuovo centroide di questo cluster
    centroids[blockIdx.x * n + threadIdx.x] = centroids[blockIdx.x * n + threadIdx.x] / dimS[blockIdx.x];
}


unsigned long parseData(ifstream &csv, vector<double> &data, vector<string> &labels) {
        double *domainMax;
        unsigned long n = 0;
        int index = 0;
        while (!csv.eof()) {
            string row;
            getline(csv, row);
            // perche ovviamente in c++ string.split() non esiste...
            vector<string> rowArr;
            const char delimiter = ';';
            //il primo token è il label del vettore
            labels.push_back(row.substr(0, row.find(delimiter)));
            //i seguenti sono le coordinate
            size_t start = row.find(delimiter) + 1;
            size_t end = row.find(delimiter, start);
            while (end != string::npos) {
                rowArr.push_back(row.substr(start, end - start));
                start = end + 1; // scansa il ';'
                end = row.find(delimiter, start);
            }
            rowArr.push_back(row.substr(start));

            if (n == 0) {
                n = rowArr.size();
                domainMax = new double[n];
                for (int i = 0; i < n; i++) {
                    domainMax[i] = std::numeric_limits<double>::lowest();
                }
            }
            if (n == rowArr.size())
            {
                for (int i = 0; i < n; i++)
                {
                    data.push_back(stod(rowArr[i]));
                    if (data[index] > domainMax[i])
                    {
                        domainMax[i] = data[index];
                    }
                    index++;
                }
            }
        }
        for(int j=0; j<data.size(); j++) {
            data[j] = data[j] / domainMax[j%n]; // normalizza i dati -> tutto è adesso fra 0 e 1
        }
        return n;
    }

void printClusters(vector<string> &labels, int clusters[], const int dimS[], size_t clusterNumber, size_t dataSize) {
    //string table = "Cluster:\n\n";
    ostringstream table;
    size_t width = min(max(labels[0].length()*5/2, 6lu), 20lu);
    table << "Clusters:\n\n";
    int processedCluster[clusterNumber];
    for (size_t col = 0; col < clusterNumber; col++) {
        table << setw(width) << col;
        processedCluster[col] = 0;
    }
    table << setw(width/2) << endl;
    for (int i = 0; i < clusterNumber*width; i++) {
        table << "·";
    }
    table << endl;
    size_t processed = 0;
    while(processed < dataSize){
        for (size_t col = 0; col < clusterNumber; col++) {
            if (dimS[col] > processedCluster[col]) {
                table << setw(width) << labels[clusters[processed]];
                processedCluster[col] += 1;
                processed++;
            } else {
                table << setw(width) << " ";
            }
        }
        table << endl;
    }
    cout << table.str() << endl;
    // write output on a file
    ofstream myfile;
    myfile.open ("output.txt");
    myfile << table.str();
    myfile.close();
}

int main(){
    double *res;
    double *sum;
    double *sum_host;
    int *S;
    int *S_host;
    int *S_host_old;
    int *dimS;
    int *dimS_host;
    int *elemLength;
    double *totalNormAvg;
    double *centroids;
    double *data_d;

    vector<double> dataVec(0);
    vector<string> dataLabel(0);
    //vector<double> totalNormAvg(CLUSTER_NUMBER);
    ifstream myfile;
//    myfile.open("../../datasetProva.csv");
    myfile.open("../../test_reale.csv");
    unsigned long n = parseData(myfile, dataVec, dataLabel);
    myfile.close();
    double data[dataVec.size()];
    std::copy(dataVec.begin(), dataVec.end(), data);
    cout << "n = " << n << "\n";
    cout << "Datavec size: " << dataVec.size() << endl;
    cout << "Data size: " << ARRAYSIZEOF(data)/n << endl;

    // Allocate host memory
    S_host=(int*)malloc(sizeof(int) * dataVec.size()/n);
    S_host_old=(int*)malloc(sizeof(int) * dataVec.size()/n);
    dimS_host=(int*)malloc(sizeof(int) * CLUSTER_NUMBER);
    sum_host = (double*)malloc(sizeof(double) * dataVec.size()/n*CLUSTER_NUMBER);

    // Allocate device memory
    hipMalloc((void**)&res, sizeof(double) * dataVec.size()*CLUSTER_NUMBER);
    hipMalloc((void**)&sum, sizeof(double) * dataVec.size()/n*CLUSTER_NUMBER);
    hipMalloc((void**)&S, sizeof(int) * dataVec.size()/n);
    hipMalloc((void**)&dimS, sizeof(double) * CLUSTER_NUMBER);
    hipMalloc((void**)&totalNormAvg, sizeof(double) * CLUSTER_NUMBER);
    hipMalloc((void**)&elemLength, sizeof(int));
    hipMalloc((void**)&centroids, sizeof(double) * CLUSTER_NUMBER*n);
    hipMalloc((void**)&data_d, sizeof(double) * dataVec.size());

    // Transfer data from host to device memory
    hipMemcpy(data_d, data, sizeof(double) * dataVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(elemLength, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(centroids, data, sizeof(double)*n*CLUSTER_NUMBER, hipMemcpyHostToDevice); //i primi CLUSTER_NUMBER vettori di data per provare

    // Executing kernel
    size_t iterazioni = 0;
    bool newClusterDifferent = true;
    while(newClusterDifferent){
        kmeanDevice<<<1,1>>>(S, dimS, n, totalNormAvg,  data_d, centroids, res, sum, ARRAYSIZEOF(data)/n, CLUSTER_NUMBER);
        hipDeviceSynchronize();
        meanz<<<CLUSTER_NUMBER, n>>>(centroids, data_d, S, dimS, n);
        hipDeviceSynchronize();
        hipMemcpy(S_host, S, sizeof(int) * dataVec.size()/n, hipMemcpyDeviceToHost);
        for(int i=0;i<dataVec.size()/n;i++){
            if(S_host[i]!= S_host_old[i]){
                newClusterDifferent = true;
                break;
            }else{
                newClusterDifferent = false;            }
        }
        int *tmp = S_host_old;
        S_host_old = S_host;
        S_host = tmp;
        iterazioni++;
    }
    
//    kmeanDevice<<<1,1>>>(S, dimS, n, totalNormAvg,  data_d, centroids, res, sum, ARRAYSIZEOF(data)/n, CLUSTER_NUMBER);
//    hipDeviceSynchronize();
//    meanz<<<CLUSTER_NUMBER, n>>>(centroids, data_d, S, dimS, n);
//    hipDeviceSynchronize();

    // Transfer data back to host memory
    hipMemcpy(S_host, S, sizeof(int) * dataVec.size()/n, hipMemcpyDeviceToHost);
    hipMemcpy(dimS_host, dimS, sizeof(int) * CLUSTER_NUMBER, hipMemcpyDeviceToHost);
    cout << "Dimensione grid: " << CLUSTER_NUMBER << "x" << ARRAYSIZEOF(data)/n << endl;
    cout << "Dimensioni dei cluster\n";
    for(int i = 0; i<CLUSTER_NUMBER; i++){
        cout << dimS_host[i] << endl;
    }
    cout << "\n";
    printClusters(dataLabel, S_host, dimS_host, CLUSTER_NUMBER, ARRAYSIZEOF(data)/n);

    // Verification

    // Deallocate device memory
    hipFree(res);
    hipFree(sum);
    hipFree(S);
    hipFree(dimS);
    hipFree(totalNormAvg);
    hipFree(elemLength);
    hipFree(centroids);
    hipFree(data_d);

    // Deallocate host memory
    free(S_host);
    free(S_host_old);
    free(dimS_host);
    free(sum_host);

    cout << "Esecuzione terminata in " << iterazioni << " iterazioni." << endl;
}


