#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iostream>
#include <string>
#include <fstream>
#include <utility>
#include <vector>
#include <unordered_map>
#include <future>
#include <filesystem>
#include <thread>
#include <float.h>

namespace fs = std::filesystem;

using namespace std;

#define N 10000000
#define MAX_ERR 1e-6

#define CLUSTER_NUMBER 2

__global__ void normA(double a[], double b[], double res[], int n, double sum[]) {
    res[blockIdx.x*n + threadIdx.x] = pow(a[blockIdx.x*n + threadIdx.x] - b[blockIdx.x*n + threadIdx.x], 2);
    __syncthreads();
    if(threadIdx.x == 0){
        for(int i=0; i<n;i++){
            sum[blockIdx.x] = sum[blockIdx.x] + res[blockIdx.x*n + i];
        }
    }
}

__global__ void meanz(double means[], double S[], int dimS[], int * elemLengthPtr) {// calcola centroidi
    int elemLength = *elemLengthPtr;
    means[blockIdx.x*elemLength + threadIdx.x] = 0;
    int dimSum = 0;
    // calcola la coordinata iniziale del primo vettore del cluster blockIdx.x
    for (int j=0; j<blockIdx.x; j++) {
        dimSum += dimS[j];
    }
    dimSum = dimSum*elemLength;
    // scorre tutti gli elementi del cluster (la grandezza del cluster e' in dimS[blockIdx.x])
    for (int i=0; i<dimS[blockIdx.x]; i++) {
        dimSum += elemLength;
        // quindi alla fine in means c'e' la somma di tutte le n-esime coordinate di ogni elemento del cluster
        means[blockIdx.x *elemLength + threadIdx.x] = means[blockIdx.x *elemLength + threadIdx.x] + S[dimSum + threadIdx.x];

    }
    // divide per la dimensione del cluster per fare la media -> coordinata n-esima del nuovo centroide di questo cluster
    means[blockIdx.x *elemLength + threadIdx.x] = means[blockIdx.x *elemLength + threadIdx.x] / dimS[blockIdx.x];
}

__global__ string getCharAlph(int i) {
     return "ABCDEFGHIJKLMNOPQRSTUVWXYZ"[i-1];
}

__global__ void kmean(double totalNormAvg[], double entry[][5], double means[]) {
        double norm = 0;
        string S [][];// dimensione clusterNumber. S e' array list di array list
        /*for (int j = 0; j < S.length; j++) {
            S[j] = new ArrayList<>();
        }*/
        for (int h = 0; h < totalNormAvg.length; h++) {// array delle norme 
            totalNormAvg[h] = 0;
        }
        for (int e = 0 ; e < entry.length; e++) {
            int posMin = 0;

            double min = DBL_MAX;
            for (int h = 0; h < means.length; h++) {
                //double norm = norm(entry.getValue(), means[h]);
                if (norm < min) {
                    min = norm;
                    posMin = h;
                }
            }
            string key = getCharAlph(e);
            S[posMin][0]=key; //è sbagliato era solo per provare
            totalNormAvg[posMin] = totalNormAvg[posMin] + min;
        }
        for (int i = 0; i < totalNormAvg.length; i++) {
            if (S[i].length > 0) {
                totalNormAvg[i] = totalNormAvg[i] / S[i].length;
            }
        }
}



unsigned long parseData(ifstream &csv, vector<double> &data) {
        double *domainMax;
        unsigned long n = -1;
        int index = 0;
        while (!csv.eof()) {
            string row;
            getline(csv, row);
            //cout << row << endl;
            istringstream iss(row);
            // perche ovviamente in c++ string.split() non esiste...
            vector<string> rowArr;
            const char delimiter = ';';
            // evita il primo token, tanto è il nome del primo vettore
            int start = row.find(delimiter) + 1;
            int end = row.find(delimiter, start);
            while (end != -1) {
                rowArr.push_back(row.substr(start, end - start));
                start = end + 1; // scansa il ';'
                end = row.find(delimiter, start);
            }
            rowArr.push_back(row.substr(start));

            if (n == -1) {
                n = rowArr.size();
                domainMax = new double[n];
                for (int i = 0; i < n; i++) {
                    domainMax[i] = std::numeric_limits<double>::lowest();
                }
            }
            if (n == rowArr.size())
            {
                for (int i = 0; i < n; i++)
                {
                    data.push_back(stod(rowArr[i]));
                    if (data[index] > domainMax[i])
                    {
                        domainMax[i] = data[index];
                    }
                    index++;
                }
            }
        }
        for(int j=0; j<data.size(); j++) {
            data[j] = data[j] / domainMax[j%n]; // normalizza i dati -> tutto è adesso fra 0 e 1
        }
        return n;
    }

int main(){
    double *a, *b, *out;
    double *d_a, *d_b, *d_out;
    double *res;
    double *sum;
    double *means;
    double *S;
    int *dimS;
    int *elemLength;
    double *data_d;

    vector<double> dataVec(0);
    vector<double> totalNormAvg(CLUSTER_NUMBER);
    string s;
    ifstream myfile;
    myfile.open("../../datasetProva.csv");
    unsigned long n = parseData(myfile, dataVec);
    myfile.close();
    double data[dataVec.size()];
    std::copy(dataVec.begin(), dataVec.end(), data);
    //printf(dataVec.size());
    cout << "n = " << n << "\n";
    cout << "Data size: " << dataVec.size() << endl;

    // Allocate host memory
    a   = (double*)malloc(sizeof(double) * N);
    b   = (double*)malloc(sizeof(double) * N);
    out = (double*)malloc(sizeof(double) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&res, sizeof(double) * 10);
    hipMalloc((void**)&sum, sizeof(double) * 2);
    hipMalloc((void**)&means, sizeof(double) * 2);
    hipMalloc((void**)&S, sizeof(double) * 2);
    hipMalloc((void**)&dimS, sizeof(double) * 2);
    hipMalloc((void**)&elemLength, sizeof(int));
    //hipMalloc((void**)&data_d, sizeof(double) * dataVec.size());


    hipMalloc((void**)&d_a, sizeof(double) * N);
    hipMalloc((void**)&d_b, sizeof(double) * N);
    hipMalloc((void**)&d_out, sizeof(double) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    normA<<<CLUSTER_NUMBER,5>>>(d_a, d_b, res, 5, sum);

    hipDeviceSynchronize();

    meanz<<<CLUSTER_NUMBER, 5>>>(means, S, dimS, elemLength);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(double) * N, hipMemcpyDeviceToHost);

    // Verification

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a);
    free(b);
    free(out);

    cout << "Esecuzione terminata." << endl;
}


